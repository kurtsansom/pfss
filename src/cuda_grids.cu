#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "src/grids.h"
#include "src/ellipticalGrid.h"
#include "src/cuda_interface.h"


__global__ void
__launch_bounds__(512, 1)
kernel_print(SphericalGrid *d_grid)
{
	printf("lowerR: %E\n", d_grid->lowerR);
}


void SphericalGrid::initNULL_GPU()
{
    d_memStruct	= NULL;
    onDevice    = false;

    checkCUDAerror("SphericalGrid::initNULL_GPU");
}

void SphericalGrid::clear_GPU()
{
    if(d_memStruct != NULL)
    	hipFree(d_memStruct);

    checkCUDAerror("SphericalGrid::clear_GPU");

    onDevice = false;
    initNULL_GPU();
}

/*! this is only responsible for initializing the pointers of the object in GPU memory
 *
 */
void SphericalGrid::initGPUmemStruct()
{
	long unsigned index 			= sizeof(SphericalGrid);
	long unsigned newCudaMemAddr 	= (long unsigned)d_memStruct + index;

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->pos), &newCudaMemAddr, sizeof(Vec3D *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Vec3D);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->B), &newCudaMemAddr, sizeof(Vec3D *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Vec3D);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->psi), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->relError), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->temp), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ijk), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_imjk), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ipjk), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ijmk), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ijpk), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ijkm), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ijkp), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_imjmk), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_imjpk), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ipjmk), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ipjpk), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_imjkm), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_imjkp), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ipjkm), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ipjkp), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ijmkm), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ijmkp), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ijpkm), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ijpkp), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->g), &newCudaMemAddr, sizeof(Matrix3x3 *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->h), &newCudaMemAddr, sizeof(Matrix3x3 *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->p), &newCudaMemAddr, sizeof(Matrix3x3 *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->p_r), &newCudaMemAddr, sizeof(Matrix3x3 *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->p_t), &newCudaMemAddr, sizeof(Matrix3x3 *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->p_p), &newCudaMemAddr, sizeof(Matrix3x3 *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	checkCUDAerror("SphericalGrid::initGPUmemStruct");
}

/*! Allocates GPU memory for object. Amount of memory is (runtime) constant. All pointers to arrays / objects are
 *  initialized to point to their reserved memory addresses
 */
bool SphericalGrid::allocateGPUmem()
{
	clear_GPU();

	//				= Memory Structure		+ 					pos / B 			+ psi / relError / temp + g, h, p, p_r, p_t, p_p
	//																				+ 19 scale factors
	size_t memsize 	= sizeof(SphericalGrid) + numGridPoints * ( 2 * sizeof(Vec3D) 	+ 22 * sizeof(hcFloat) 	+ 6 * sizeof(Matrix3x3));
	SphericalGrid empty;
	hipMalloc((void **) &d_memStruct, memsize);
	hipMemcpy((char *)d_memStruct, &empty, sizeof(SphericalGrid), hipMemcpyHostToDevice);
	//initGPUmemStruct();						// TODO: unnecessary due to pushToGPU?

	checkCUDAerror("SphericalGrid::allocateGPUmem");

	return true;
}

bool SphericalGrid::pushToGPU(){

	if(numGridPoints == 0)
	{
		printf("ERROR! SphericalGrid::pushToDevice: numGridPoints=0 -> nothing to do!\n");
		return false;
	}

    allocateGPUmem();

	long unsigned index 			= 0;
	long unsigned newCudaMemAddr 	= (long unsigned)d_memStruct + index;

	hipMemcpy((char *)newCudaMemAddr, this, sizeof(SphericalGrid), hipMemcpyHostToDevice);
	initGPUmemStruct();

	index = sizeof(SphericalGrid);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, pos, numGridPoints * sizeof(Vec3D), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Vec3D);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, B, numGridPoints * sizeof(Vec3D), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Vec3D);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, psi, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, relError, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, temp, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ijk, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_imjk, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ipjk, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ijmk, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ijpk, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ijkm, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ijkp, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_imjmk, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_imjpk, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ipjmk, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ipjpk, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_imjkm, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_imjkp, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ipjkm, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ipjkp, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ijmkm, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ijmkp, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ijpkm, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ijpkp, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, g, numGridPoints * sizeof(Matrix3x3), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, h, numGridPoints * sizeof(Matrix3x3), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, p, numGridPoints * sizeof(Matrix3x3), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, p_r, numGridPoints * sizeof(Matrix3x3), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, p_t, numGridPoints * sizeof(Matrix3x3), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, p_p, numGridPoints * sizeof(Matrix3x3), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	onDevice = true;

	checkCUDAerror("SphericalGrid::pushToGPU");

	return true;
}

bool SphericalGrid::pullFromGPU()
{
	if(!onDevice)
		return false;

	clear_CPU();

	SphericalGrid *tAlloc = new SphericalGrid;
	hipMemcpy(tAlloc, d_memStruct, sizeof(SphericalGrid), hipMemcpyDeviceToHost);

	init(tAlloc->sinLatGrid, tAlloc->maxSinLat, tAlloc->minSinLat, tAlloc->lowerR, tAlloc->upperR, tAlloc->numR, false, 1.0);

	hipMemcpy(this->pos, 		tAlloc->pos, 		numGridPoints * sizeof(Vec3D), 	hipMemcpyDeviceToHost);
	hipMemcpy(this->B, 		tAlloc->B, 			numGridPoints * sizeof(Vec3D), 	hipMemcpyDeviceToHost);
	hipMemcpy(this->psi, 		tAlloc->psi, 		numGridPoints * sizeof(hcFloat), hipMemcpyDeviceToHost);
	hipMemcpy(this->relError, 	tAlloc->relError, 	numGridPoints * sizeof(hcFloat), hipMemcpyDeviceToHost);
	hipMemcpy(this->temp, 		tAlloc->temp, 		numGridPoints * sizeof(hcFloat), hipMemcpyDeviceToHost);
	// the following should not be necessary to fetch from GPU
	/*
	hipMemcpy(this->g, 		tAlloc->g,	 		numGridPoints * sizeof(Matrix3x3), hipMemcpyDeviceToHost);
	hipMemcpy(this->h, 		tAlloc->h,	 		numGridPoints * sizeof(Matrix3x3), hipMemcpyDeviceToHost);
	hipMemcpy(this->p, 		tAlloc->p,	 		numGridPoints * sizeof(Matrix3x3), hipMemcpyDeviceToHost);
	hipMemcpy(this->p_r, 		tAlloc->p_r, 		numGridPoints * sizeof(Matrix3x3), hipMemcpyDeviceToHost);
	hipMemcpy(this->p_t, 		tAlloc->p_t, 		numGridPoints * sizeof(Matrix3x3), hipMemcpyDeviceToHost);
	hipMemcpy(this->p_p, 		tAlloc->p_p, 		numGridPoints * sizeof(Matrix3x3), hipMemcpyDeviceToHost);//*/

	tAlloc->initNULL();
	delete tAlloc;

	checkCUDAerror("CudaAllocator::pullFromGPU");

	return true;
}

void SphericalGrid::extract_relError(){

	SphericalGrid *tAlloc = new SphericalGrid;;
	hipMemcpy(tAlloc, d_memStruct, sizeof(SphericalGrid), hipMemcpyDeviceToHost);

	hipMemcpy(relError, tAlloc->relError, numGridPoints * sizeof(hcFloat), hipMemcpyDeviceToHost);

	tAlloc->initNULL();
	delete tAlloc;

	checkCUDAerror("SphericalGrid::extractRelError");
}


/*! this is only responsible for initializing the pointers of the object in GPU memory
 *	TODO: so much code duplication
 */
void EllipticalGrid::initGPUmemStruct(){

	long unsigned index 			= sizeof(EllipticalGrid);
	long unsigned newCudaMemAddr 	= (long unsigned)d_memStruct + index;

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(((EllipticalGrid*)d_memStruct)->pos), &newCudaMemAddr, sizeof(Vec3D *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Vec3D); // TODO Vec3D* ?

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(((EllipticalGrid*)d_memStruct)->B), &newCudaMemAddr, sizeof(Vec3D *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Vec3D);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(((EllipticalGrid*)d_memStruct)->psi), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(((EllipticalGrid*)d_memStruct)->relError), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(((EllipticalGrid*)d_memStruct)->temp), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ijk), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_imjk), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ipjk), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ijmk), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ijpk), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ijkm), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ijkp), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_imjmk), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_imjpk), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ipjmk), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ipjpk), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_imjkm), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_imjkp), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ipjkm), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ipjkp), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ijmkm), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ijmkp), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ijpkm), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(d_memStruct->s_ijpkp), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(((EllipticalGrid*)d_memStruct)->g), &newCudaMemAddr, sizeof(Matrix3x3 *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(((EllipticalGrid*)d_memStruct)->h), &newCudaMemAddr, sizeof(Matrix3x3 *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(((EllipticalGrid*)d_memStruct)->p), &newCudaMemAddr, sizeof(Matrix3x3 *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(((EllipticalGrid*)d_memStruct)->p_r), &newCudaMemAddr, sizeof(Matrix3x3 *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(((EllipticalGrid*)d_memStruct)->p_t), &newCudaMemAddr, sizeof(Matrix3x3 *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(((EllipticalGrid*)d_memStruct)->p_p), &newCudaMemAddr, sizeof(Matrix3x3 *), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy(&(((EllipticalGrid*)d_memStruct)->a), &newCudaMemAddr, sizeof(hcFloat *), hipMemcpyHostToDevice);
	index += numR * sizeof(hcFloat);

	checkCUDAerror("EllipticalGrid::initGPUmemStruct");
}

/*! Allocates GPU memory for object. Amount of memory is (runtime) constant. All pointers to arrays / objects are
 *  initialized to point to their reserved memory addresses
 *
 */
bool EllipticalGrid::allocateGPUmem(){

	clear_GPU();

	//				= Memory Structure		 + 					pos / B 			+ psi / relError / temp	+ g,h,p,p_r,p_t,p_p			+ a
	//																				+ 19 scale factors
	size_t memsize 	= sizeof(EllipticalGrid) + numGridPoints * ( 2 * sizeof(Vec3D) 	+ 22 * sizeof(hcFloat) 	+ 6 * sizeof(Matrix3x3))	+ numR * sizeof(hcFloat);
	EllipticalGrid empty;
	hipMalloc((void **) &d_memStruct, memsize);
	hipMemcpy((char *)d_memStruct, &empty, sizeof(EllipticalGrid), hipMemcpyHostToDevice);

	checkCUDAerror("EllipticalGrid::allocateGPUmem");

	return true;
}

bool EllipticalGrid::pushToGPU(){

	if(numGridPoints == 0)
	{
		printf("ERROR! EllipticalGrid::pushToDevice: numGridPoints=0 -> nothing to do!\n");
		return false;
	}

    allocateGPUmem();

	long unsigned index 			= 0;
	long unsigned newCudaMemAddr 	= (long unsigned)d_memStruct + index;

	hipMemcpy((char *)newCudaMemAddr, this, sizeof(EllipticalGrid), hipMemcpyHostToDevice);
	initGPUmemStruct();

	index = sizeof(EllipticalGrid);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, pos, numGridPoints * sizeof(Vec3D), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Vec3D);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, B, numGridPoints * sizeof(Vec3D), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Vec3D);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, psi, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, relError, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, temp, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ijk, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_imjk, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ipjk, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ijmk, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ijpk, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ijkm, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ijkp, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_imjmk, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_imjpk, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ipjmk, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ipjpk, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_imjkm, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_imjkp, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ipjkm, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ipjkp, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ijmkm, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ijmkp, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ijpkm, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, s_ijpkp, numGridPoints * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(hcFloat);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, g, numGridPoints * sizeof(Matrix3x3), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, h, numGridPoints * sizeof(Matrix3x3), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, p, numGridPoints * sizeof(Matrix3x3), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, p_r, numGridPoints * sizeof(Matrix3x3), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, p_t, numGridPoints * sizeof(Matrix3x3), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, p_p, numGridPoints * sizeof(Matrix3x3), hipMemcpyHostToDevice);
	index += numGridPoints * sizeof(Matrix3x3);

	newCudaMemAddr = (long unsigned)d_memStruct + index;
	hipMemcpy((char *)newCudaMemAddr, a, numR * sizeof(hcFloat), hipMemcpyHostToDevice);
	index += numR * sizeof(hcFloat);

	onDevice = true;

	checkCUDAerror("EllipticalGrid::pushToGPU");

	return true;
}

bool EllipticalGrid::pullFromGPU()
{
	if(!onDevice)
		return false;

	clear_CPU();

	EllipticalGrid *tAlloc = new EllipticalGrid;
	hipMemcpy(tAlloc, d_memStruct, sizeof(EllipticalGrid), hipMemcpyDeviceToHost);

	init(tAlloc->sinLatGrid, tAlloc->maxSinLat, tAlloc->minSinLat, tAlloc->lowerR, tAlloc->upperR,	tAlloc->numR, false, 1.0);

	hipMemcpy(this->pos, 		tAlloc->pos, 		numGridPoints * sizeof(Vec3D), 		hipMemcpyDeviceToHost);
	hipMemcpy(this->B, 		tAlloc->B, 			numGridPoints * sizeof(Vec3D), 		hipMemcpyDeviceToHost);
	hipMemcpy(this->psi, 		tAlloc->psi, 		numGridPoints * sizeof(hcFloat), 	hipMemcpyDeviceToHost);
	hipMemcpy(this->relError, 	tAlloc->relError, 	numGridPoints * sizeof(hcFloat), 	hipMemcpyDeviceToHost);
	hipMemcpy(this->temp, 		tAlloc->temp, 		numGridPoints * sizeof(hcFloat), 	hipMemcpyDeviceToHost);
	// the following should not be necessary to pull from GPU
	/*
	hipMemcpy(this->g, 		tAlloc->g, 			numGridPoints * sizeof(Matrix3x3), 	hipMemcpyDeviceToHost);
	hipMemcpy(this->h, 		tAlloc->h, 			numGridPoints * sizeof(Matrix3x3), 	hipMemcpyDeviceToHost);
	hipMemcpy(this->p, 		tAlloc->p, 			numGridPoints * sizeof(Matrix3x3), 	hipMemcpyDeviceToHost);
	hipMemcpy(this->p_r, 		tAlloc->p_r,		numGridPoints * sizeof(Matrix3x3), 	hipMemcpyDeviceToHost);
	hipMemcpy(this->p_t, 		tAlloc->p_t,		numGridPoints * sizeof(Matrix3x3), 	hipMemcpyDeviceToHost);
	hipMemcpy(this->p_p, 		tAlloc->p_p,		numGridPoints * sizeof(Matrix3x3), 	hipMemcpyDeviceToHost);//*/
	hipMemcpy(this->a, 		tAlloc->a, 			numR * sizeof(hcFloat), 			hipMemcpyDeviceToHost);

	tAlloc->initNULL();
	delete tAlloc;
	checkCUDAerror("EllipticalGrid::pullFromGPU");

	return true;
}

void EllipticalGrid::extract_relError()
{
	EllipticalGrid *tAlloc = new EllipticalGrid;;
	hipMemcpy(tAlloc, d_memStruct, sizeof(EllipticalGrid), hipMemcpyDeviceToHost);

	hipMemcpy(relError, tAlloc->relError, numGridPoints * sizeof(hcFloat), hipMemcpyDeviceToHost);

	tAlloc->initNULL();
	delete tAlloc;

	checkCUDAerror("EllipticalGrid::extract_relError");
}
